#define gpuErrorCheck(ans)                    \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
#include <stdlib.h>
#include <algorithm>
#include <cmath>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "gpu.cuh"
#include "defines.h"
#include "hip/hip_runtime_api.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

float solveWithGpu()
{
    return 0;
}
