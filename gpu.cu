#include "hip/hip_runtime.h"
#define gpuErrorCheck(ans)                    \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
#include <stdlib.h>
#include <algorithm>
#include <cmath>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "gpu.cuh"
#include "defines.h"
#include "hip/hip_runtime_api.h"
#include <vector>
using namespace std;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void findHammingDistance(int *d_bitSequences, int *d_output, int *d_flag, int vectorCount, int vectorLength)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= vectorCount)
        return;
    for (int i = idx + 1; i < vectorCount; i++)
    {
        int hammingDistance = 0;
        for (int j = 0; j < vectorLength; j++)
        {
            if ((d_bitSequences[idx * vectorLength + j] != d_bitSequences[i * vectorLength + j]))
            {
                hammingDistance++;
            }
        }
        if (hammingDistance == 1)
        {
            int pairIdx = min(idx, i);
            if (!d_flag[pairIdx])
            {
                atomicCAS(&d_output[pairIdx], -1, max(idx, i));
                d_flag[pairIdx] = 1;
            }
        }
    }
}

vector<pair<string, string>> solveWithGpu(vector<string> bitSequences)
{
    vector<pair<string, string>> pairs;

    int vectorCount = bitSequences.size();
    int vectorLength = bitSequences[0].length();

    int *h_bitSequences = new int[vectorCount * vectorLength];
    int *h_output = new int[vectorCount];
    int *h_flag = new int[vectorCount];
    int *d_bitSequences, *d_output, *d_flag;

    for (int i = 0; i < vectorCount; i++)
    {
        for (int j = 0; j < vectorLength; j++)
        {
            h_bitSequences[i * vectorLength + j] = bitSequences[i][j] - '0';
        }
        h_output[i] = -1;
        h_flag[i] = 0;
    }

    hipMalloc(&d_bitSequences, vectorCount * vectorLength * sizeof(int));
    hipMalloc(&d_output, vectorCount * sizeof(int));
    hipMalloc(&d_flag, vectorCount * sizeof(int));
    hipMemcpy(d_bitSequences, h_bitSequences, vectorCount * vectorLength * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_output, h_output, vectorCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_flag, h_flag, vectorCount * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (vectorCount + blockSize - 1) / blockSize;
    findHammingDistance<<<numBlocks, blockSize>>>(d_bitSequences, d_output, d_flag, vectorCount, vectorLength);
    hipMemcpy(h_output, d_output, vectorCount * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_flag, d_flag, vectorCount * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < vectorCount; i++)
    {
        if (h_output[i] != -1)
        {
            pairs.push_back({bitSequences[i], bitSequences[h_output[i]]});
        }
    }

    hipFree(d_bitSequences);
    hipFree(d_output);
    hipFree(d_flag);
    delete[] h_bitSequences;
    delete[] h_output;
    delete[] h_flag;

    return pairs;
}