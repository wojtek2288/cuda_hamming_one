#include "hip/hip_runtime.h"
#define gpuErrorCheck(ans)                    \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
#include <stdlib.h>
#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>

#include "hip/hip_runtime.h"
#include ""
#include "gpu.cuh"
#include "defines.h"
#include "hip/hip_runtime_api.h"
using namespace std;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

// https://stackoverflow.com/questions/27086195/linear-index-upper-triangular-matrix
__global__ void findPairs(int *d_bitSequences, int *pairs, int n, int len)
{
    unsigned long long int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < n * (n - 1) / 2)
    {
        unsigned long long int i = n - 2 - floor(sqrt((float)(-8 * k + 4 * n * (n - 1) - 7)) / 2.0 - 0.5);
        unsigned long long int j = k + i + 1 - n * (n - 1) / 2 + (n - i) * ((n - i) - 1) / 2;
        i = j - i - 1;

        int hammingDistance = 0;

        for (int l = 0; l < len; l++)
        {
            hammingDistance += __popc(d_bitSequences[i * len + l] ^ d_bitSequences[j * len + l]);
            if (hammingDistance > 1)
            {
                break;
            }
        }

        if (hammingDistance == 1)
        {
            atomicAdd(pairs, 1);
        }
    }
}

int solveWithGpu(vector<string> bitSequences)
{
    int vectorCount = bitSequences.size();
    int vectorLength = bitSequences[0].length();

    int *h_bitSequences = new int[vectorCount * vectorLength];
    int *h_pairs;
    int *d_bitSequences, *d_pairs;

    for (int i = 0; i < vectorCount; i++)
    {
        for (int j = 0; j < vectorLength; j++)
        {
            h_bitSequences[i * vectorLength + j] = bitSequences[i][j] - '0';
        }
    }

    gpuErrorCheck(hipMalloc(&d_bitSequences, vectorCount * vectorLength * sizeof(int)));
    gpuErrorCheck(hipMalloc(&d_pairs, sizeof(int)));
    gpuErrorCheck(hipMemcpy(d_bitSequences, h_bitSequences, vectorCount * vectorLength * sizeof(int), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemset(d_pairs, 0, sizeof(int)));

    int threadCount = 512;
    long long unsigned int n = vectorCount * (vectorCount - 1) / 2;
    int blockCount = (n + threadCount - 1) / threadCount + 1;

    findPairs<<<blockCount, threadCount>>>(d_bitSequences, d_pairs, vectorCount, vectorLength);

    gpuErrorCheck(hipMemcpy(h_pairs, d_pairs, sizeof(int), hipMemcpyDeviceToHost));
    gpuErrorCheck(hipFree(d_pairs));
    gpuErrorCheck(hipFree(d_bitSequences));

    delete[] h_bitSequences;

    return *h_pairs;
}